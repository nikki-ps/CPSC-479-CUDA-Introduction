#include <stdio.h>
#include <hip/hip_runtime.h>
//kernel to initialize array with all zeros
__global__ void dkernel(unsigned *array, int size) { 
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x; 
  if (id < size)  array[id] = 0; 
} 
//kernel to initialize array[i] = i
__global__ void bkernel(unsigned *array, int size) {
    unsigned id = blockIdx.x *blockDim.x + threadIdx.x;
    if (id < size) {
        array[id] = id;
    }
}
#define BLOCKSIZE 1024 
int main(void) { 
    //problem 1
    //Initializing an array of size 32 to all zeros in parallel.
    unsigned size = 32;
    unsigned *arr, *harr;
    hipMalloc(&arr, size * sizeof(unsigned));
    harr = (unsigned *)malloc(size * sizeof(unsigned)); 
    unsigned nblocks = ceil((float) size / BLOCKSIZE); 
    printf("Created array size %d and filled with 0.\n", size); 
    dkernel<<<nblocks, BLOCKSIZE>>>(arr, size);
    hipMemcpy(harr, arr, size * sizeof(unsigned), hipMemcpyDeviceToHost);
    //For loop used to print array in order to checking contents
    /*for (unsigned ii = 0; ii < size; ++ii) 
    { 
        printf("%4d ", harr[ii]); 
    } */

    //problem 2
    //changing the size of the array to 1024
    size = 1024;
    hipMalloc(&arr, size * sizeof(unsigned));
    harr = (unsigned *)malloc(size * sizeof(unsigned)); 
    nblocks = ceil((float) size / BLOCKSIZE); 
    printf("Created array size %d and filled with 0.\n", size); 
    dkernel<<<nblocks, BLOCKSIZE>>>(arr, size);
    hipMemcpy(harr, arr, size * sizeof(unsigned), hipMemcpyDeviceToHost); 
    //problem 3
    //using another kernel to add i to array[i]
    bkernel<<<nblocks, BLOCKSIZE>>>(arr, size);
    hipMemcpy(harr, arr, size * sizeof(unsigned), hipMemcpyDeviceToHost); 
    printf("Created array size %d and filled with array[i] = i.\n", size); 
    //For loop used to print array in order to checking contents
    /* for (unsigned ii = 0; ii < size; ++ii) { 
        printf("%4d ", harr[ii]); 
    } */

    //problem 4
    //changing the array to size 8000 and adding i to array[i]
    size = 8000;
    hipMalloc(&arr, size * sizeof(unsigned));
    harr = (unsigned *)malloc(size * sizeof(unsigned)); 
    nblocks = ceil((float) size / BLOCKSIZE); 
    printf("Created array size %d and filled with array[i] = i.\n", size); 
    bkernel<<<nblocks, BLOCKSIZE>>>(arr, size);
    hipMemcpy(harr, arr, size * sizeof(unsigned), hipMemcpyDeviceToHost); 
    //For loop used to print array in order to checking contents
    /* for (unsigned ii = 0; ii < size; ++ii) 
    { 
        printf("%4d ", harr[ii]); 
    } */
} 